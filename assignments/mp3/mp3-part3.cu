#include "hip/hip_runtime.h"
// In Part 3, we'll use the stream compaction implementation we built
// in Part 2 to create a simple schedule for the second round of
// Black-Scholes jobs from Part 1. Since subsequent rounds of our options
// pricing are very sparse, a naive scheduling of this job will
// cause the majority of CUDA threads to idle while ther active warp
// neighbors work on stocks from the first round which meet our criteria. A better
// scheduling should compact all of the data which meets or exceeds the
// threshold for subsequent rounds to the front of the input, and launch only as many
// threads as there is work to do. This will ensure that all of the CUDA threads
// launched in subsequent rounds will be active (modulo the few inactive threads
// at the end of the last block). This compaction scheme eliminates the warp
// divergence penalty and increases the throughput of the second round of Black-Scholes.

// Note that your stream compaction implementation may come with significant overhead.
// How many sparse rounds of Black-Scholes does it take before the cost becomes worth
// it? It may take quite a few when computing in single precision. For double precision,
// it requires just a handful. You can control whether the "real" type is "float"
// (single precision) or "double" (double precision) by editing the Makefile.

// If you've made it this far, Part 3 should be trivial. The only thing you need
// to do is launch kernels and, as always, allocate & deallocate device storage in
// mp3-part3.cu and implement the body of compact_options in compact.cu. This is mostly
// a copy and paste job from compact_even_elements, but this time, there is more data
// to move around. Search for the lines marked TODO: and get hacking!

#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <iostream>
#include <limits>
#include <numeric>

#include "black_scholes.h"
#include "compact.h"
#include "mp3-util.h"


bool allocate_device_storage(real *&d_first_round_call_result, real *&d_first_round_put_result,
                             real *&d_subsequent_round_call_result, real *&d_subsequent_round_put_result,
                             real *&d_stock_price, real *&d_option_strike,
                             real *&d_option_years,
                             real *&d_compacted_stock_price,
                             real *&d_compacted_option_strike,
                             real *&d_compacted_option_years,
                             const size_t n)
{
  // TODO: your device memory allocations here
  // TODO: don't forget to check for CUDA errors!

  // TODO: return true to indicate successful memory allocation
  return false;
}


void deallocate_device_storage(real *d_first_round_call_result, real *d_first_round_put_result,
                               real *d_subsequent_round_call_result, real *d_subsequent_round_put_result,
                               real *d_stock_price, real *d_option_strike,
                               real *d_option_years,
                               real *d_compacted_stock_price,
                               real *d_compacted_option_strike,
                               real *d_compacted_option_years)
{
  // TODO: your device memory deallocations here
  // TODO: don't forget to check for CUDA errors!
}


int main(void)
{
  event_pair timer;

  const size_t num_subsequent_rounds = 5;
  float compaction_time = 0;
  std::vector<float> gpu_time(1 + num_subsequent_rounds);
  std::vector<float> cpu_time(1 + num_subsequent_rounds);

  // create arrays for 4M options
  size_t num_options = 1<<22;

  // allocate host storage
  std::vector<real> h_first_round_call_result(num_options,0);
  std::vector<real> h_first_round_put_result(num_options, 0);
  std::vector<real> h_subsequent_round_call_result(num_options,0);
  std::vector<real> h_subsequent_round_put_result(num_options, 0);
  std::vector<real> h_stock_price(num_options);
  std::vector<real> h_option_strike(num_options);
  std::vector<real> h_option_years(num_options);

  // generate options set
  srand(5347);
  for(int i = 0; i < num_options; ++i)
  {
    h_stock_price[i]   = random_real(5.0,  30.0);
    h_option_strike[i] = random_real(1.0, 100.0);
    h_option_years[i]  = random_real(0.25, 10.0);
  }

  // some pointers to the data set which will live in device memory
  real *d_first_round_call_result      = 0;
  real *d_first_round_put_result       = 0;
  real *d_subsequent_round_call_result = 0;
  real *d_subsequent_round_put_result  = 0;
  real *d_stock_price                  = 0;
  real *d_option_strike                = 0;
  real *d_option_years                 = 0;
  real *d_compacted_stock_price        = 0;
  real *d_compacted_option_strike      = 0;
  real *d_compacted_option_years       = 0;

  // allocate device storage
  if(!allocate_device_storage(d_first_round_call_result, d_first_round_put_result,
                              d_subsequent_round_call_result, d_subsequent_round_put_result,
                              d_stock_price, d_option_strike, d_option_years,
                              d_compacted_stock_price,
                              d_compacted_option_strike,
                              d_compacted_option_years,
                              num_options))
  {
    std::cerr << "Error allocating device memory!" << std::endl;
    exit(-1);
  }

  // fill the result arrays with 0
  hipMemset(d_first_round_call_result,      0, sizeof(real) * num_options);
  hipMemset(d_first_round_put_result,       0, sizeof(real) * num_options);
  hipMemset(d_subsequent_round_call_result, 0, sizeof(real) * num_options);
  hipMemset(d_subsequent_round_put_result,  0, sizeof(real) * num_options);

  // copy input to GPU
  start_timer(&timer);
  // TODO: your host to device copies here
  stop_timer(&timer, "host to device copy of input"); 


  // BEGIN ROUND 0

  // we will use the two following parameters
  // to first round of the Black-Scholes algorithm
  const real first_round_riskless_rate = 0.02;
  const real first_round_volatility    = 0.30;

  // do one round of Black-Scholes using our parameters
  start_timer(&timer);
  // TODO: your black_scholes_kernel launch here
  gpu_time[0] = stop_timer(&timer, "GPU Black-Scholes round 0");
  check_cuda_error("GPU Black-Scholes round 0", __FILE__, __LINE__);

  // do round 0 of Black-Scholes on the host
  start_timer(&timer);
  black_scholes_host(&h_stock_price[0],
                     &h_option_strike[0],
                     &h_option_years[0],
                     &h_first_round_call_result[0],
                     &h_first_round_put_result[0],
                     first_round_riskless_rate,
                     first_round_volatility,
                     num_options);
  cpu_time[0] = stop_timer(&timer, "CPU Black-Scholes round 0");

  // validate gpu results from round 0
  std::vector<real> h_validate_me(num_options);
  hipMemcpy(&h_validate_me[0], d_first_round_call_result, sizeof(real) * num_options, hipMemcpyDeviceToHost);
  // pass true as a final optional argument to fuzzy_validate for verbose output
  if(!fuzzy_validate(&h_validate_me[0], &h_first_round_call_result[0], num_options))
  {
    std::cerr << "Error: round 0 of call results don't match!" << std::endl;
    exit(-1);
  }

  hipMemcpy(&h_validate_me[0],  d_first_round_put_result,  sizeof(real) * num_options, hipMemcpyDeviceToHost);
  if(!fuzzy_validate(&h_validate_me[0], &h_first_round_put_result[0], num_options))
  {
    std::cerr << "Error: round 0 of put results don't match!" << std::endl;
    exit(-1);
  }


  // BEGIN COMPACTION


  // in subsequent rounds, select the stocks whose call & put prices from the first round
  // meet or exceed these thresholds
  const real min_call_threshold = 2.0;
  const real min_put_threshold  = 4.0;

  // compact the options, copying those that meet our call & put thresholds
  // to the arrays for round 2
  start_timer(&timer);
  size_t num_compacted_options = 0;
  // TODO: your call to compact_options here
  compaction_time = stop_timer(&timer, "GPU Compaction");


  // BEGIN SUBSEQUENT ROUNDS

  size_t num_compacted_options_reference = 0;

  for(int round = 1; round < num_subsequent_rounds + 1; ++round)
  {
    // change the parameters of the model in each subsequent round
    const real riskless_rate = random_real(0.03, 0.04);
    const real volatility    = random_real(0.50, 0.60);

    // do round of Black-Scholes using new parameters on the device
    start_timer(&timer);
    // TODO: your black_scholes_kernel launch here
    char message[256];
    sprintf(message, "GPU Black-Scholes round %d", round);
    gpu_time[round] = stop_timer(&timer, message);
    check_cuda_error(message, __FILE__, __LINE__);


    // do a round of Black-Scholes on the host using new parameters
    // filter the set of options to compute given the results of the last round,
    // but compact the output
    start_timer(&timer);
    num_compacted_options_reference =
      compacted_black_scholes_host(&h_stock_price[0],
                                   &h_option_strike[0],
                                   &h_option_years[0],
                                   &h_first_round_call_result[0],
                                   &h_first_round_put_result[0],
                                   &h_subsequent_round_call_result[0],
                                   &h_subsequent_round_put_result[0],
                                   min_call_threshold,
                                   min_put_threshold,
                                   riskless_rate,
                                   volatility,
                                   num_options);
    sprintf(message, "CPU Black-Scholes round %d", round);
    cpu_time[round] = stop_timer(&timer, message);

    if(num_compacted_options_reference != num_compacted_options)
    {
      std::cerr << "Error: round " << round << " num_compacted_options (" << num_compacted_options << ") doesn't match num_compacted_options_reference (" << num_compacted_options_reference << ")" << std::endl;
      exit(-1);
    }

    // validate gpu results from this round
    hipMemcpy(&h_validate_me[0], d_subsequent_round_call_result, sizeof(real) * num_compacted_options_reference, hipMemcpyDeviceToHost);
    if(!fuzzy_validate(&h_validate_me[0], &h_subsequent_round_call_result[0], num_compacted_options_reference))
    {
      std::cerr << "Error: round " << round << " of call results don't match!" << std::endl;
      exit(-1);
    }

    hipMemcpy(&h_validate_me[0],  d_subsequent_round_put_result,  sizeof(real) * num_compacted_options_reference, hipMemcpyDeviceToHost);
    if(!fuzzy_validate(&h_validate_me[0], &h_subsequent_round_put_result[0], num_compacted_options_reference))
    {
      std::cerr << "Error: round " << round << " of put results don't match!" << std::endl;
      exit(-1);
    }

  } // end for subsequent round

  deallocate_device_storage(d_first_round_call_result, d_first_round_put_result,
                            d_subsequent_round_call_result, d_subsequent_round_put_result,
                            d_stock_price, d_option_strike,
                            d_option_years,
                            d_compacted_stock_price,
                            d_compacted_option_strike,
                            d_compacted_option_years);

  // output a report
  std::cout << std::endl;

  real first_round_gpu_throughput = static_cast<real>(num_options) / (gpu_time[0] / 1000.0f);
  real first_round_cpu_throughput = static_cast<real>(num_options) / (cpu_time[0] / 1000.0f);

  std::cout << "Round 0: " << num_options << " options" << std::endl;
  std::cout << "Throughput of GPU Black-Scholes Round 0: " << (first_round_gpu_throughput / 1e6) << " Megaoptions/sec" << std::endl;
  std::cout << "Throughput of CPU Black-Scholes Round 0: " << (first_round_cpu_throughput / 1e6) << " Megaoptions/sec" << std::endl;
  std::cout << "Speedup of Round 0: " << first_round_gpu_throughput / first_round_cpu_throughput << "x" << std::endl << std::endl;

  for(int i = 1; i < gpu_time.size(); ++i)
  {
    real gpu_throughput = static_cast<real>(num_compacted_options_reference) / (gpu_time[i] / 1000.0f);
    real cpu_throughput = static_cast<real>(num_compacted_options_reference) / (cpu_time[i] / 1000.0f);

    std::cout << "Round " << i << ": " << num_compacted_options_reference << " options" << std::endl;
    std::cout << "Throughput of GPU Black-Scholes Round " << i << ": " << (gpu_throughput / 1e6) << " Megaoptions/sec" << std::endl;
    std::cout << "Throughput of CPU Black-Scholes Round " << i << ": " << (cpu_throughput / 1e6) << " Megaoptions/sec" << std::endl;
    std::cout << "Speedup of Round " << i << ": " << gpu_throughput / cpu_throughput << "x" << std::endl << std::endl;
  }

  // report overall performance
  real total_gpu_time = compaction_time + std::accumulate(gpu_time.begin(), gpu_time.end(), 0.0);
  real total_cpu_time = std::accumulate(cpu_time.begin(), cpu_time.end(), 0.0);
  real gpu_throughput = static_cast<real>(num_options + num_subsequent_rounds*num_compacted_options_reference) / ((total_gpu_time) / 1000.0f);
  real cpu_throughput = static_cast<real>(num_options + num_subsequent_rounds*num_compacted_options_reference) / ((total_cpu_time) / 1000.0f);

  std::cout << "Overall GPU throughput: " << (gpu_throughput / 1e6) << " Megaoptions/sec" << std::endl;
  std::cout << "Overall CPU throughput: " << (cpu_throughput / 1e6) << " Megaoptions/sec" << std::endl << std::endl;

  std::cout << "Overall speedup: " << gpu_throughput / cpu_throughput << "x" << std::endl;

  return 0;
}

