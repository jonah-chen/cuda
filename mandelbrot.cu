
#include <hip/hip_runtime.h>
#include <iostream>

/* Generates a mandelbrot set image
 *
 * The elements of bounds is the real min/max and imaginary min/max in that order
 * iterations is the number of iterations of z^2+c on each pixel
 * The percision is the real and imaginary step size in that order
 */
__global__ 
void mandelbrot(int* image, 
    const double left, 
    const double right, 
    const double down, 
    const double up, 
    const int iterations, 
    const double real_precision, 
    const double im_precision)
{
    // Representation of the image is in row-major order. Thus, the (re,im) pixel is image[re+im*re_size]
    
    int x = threadIdx.x + blockIdx.x * blockDim.x; // real part
    int y = threadIdx.y + blockIdx.y * blockDim.y; // imaginary part

    const int re_size = (int)((right-left)/real_precision);
    const int im_size = (int)((up-down)/im_precision);

    if (x < re_size && y < im_size) // check that it is in bounds
    {
        // c=r+it is the original number
        double r = left + real_precision*x;
        double t = down + im_precision*y;

        // start from z=0
        double zr = 0.0;
        double zt = 0.0;

        for (int iteration = 0; iteration < iterations-1; ++iteration)
        {
            // perform an iteration of z^2+c
            zr = zr*zr - zt*zt + r;
            zt = zr*zt*2 + t;

            // check for |z|>2 (or |z|^2>4)
            if (zr*zr+zt*zt > 4.0)
            {
                // convert to image. here, a linear gamma curve is used
                image[x+y*re_size] = iteration;
                return;
            }
        }
        image[x+y*re_size] = iterations-1;
    }
}

int main(int argc, char* argv[])
{
    // the default bounds for the mandelbrot set is usually -2<Re(z)<1 and -1<Im(z)<1
    double bounds[4] = {-2.0, 1.0, -1.0, 1.0};
    
    // given precision of 0.001, it will generate 3000x2000 image which is about 6 megapixel
    double precision[2] = {0.001, 0.001};
    
    // i feel like making iterate 256 times
    int iterations = 256;

    int *d_image, *h_image;
    int re_size, im_size;
    
    // compute the size of the image
    re_size = (int)((bounds[1]-bounds[0])/precision[0]);
    im_size = (int)((bounds[3]-bounds[2])/precision[1]);
    
    // allocate the GPU memory for the image
    if (hipMalloc(&d_image, re_size*im_size*sizeof(int)) != hipSuccess) 
    {
        std::cout << "The device does not have enough memory. Program exited with error code -1." << std::endl;
        return -1;
    }

    // set the memory to 0
    if (hipMemset(d_image, 0, re_size*im_size*sizeof(int)) != hipSuccess)
    {
        std::cout << "Failed to set memory to 0. Program exited with error code -2." << std::endl;
        return -2;
    }

    // create the grid and blocks to call the method. 32x32=1024 threads.
    dim3 grid((re_size+31)/32, (im_size+31)/32, 1);
    dim3 block(32,32,1);
    mandelbrot <<<grid, block>>> (d_image, bounds[0], bounds[1], bounds[2], bounds[3], iterations, precision[0], precision[1]);
    hipDeviceSynchronize();

    h_image = (int*)calloc(re_size*im_size, sizeof(int));
    hipMemcpy(h_image, d_image, re_size*im_size*sizeof(int), hipMemcpyDeviceToHost);

    std::cout << h_image[0+0*re_size] << std::endl;
    
    return 0;
}